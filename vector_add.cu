
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<assert.h>
#include<time.h>
#define N 15000000
#define FILENAME "vector_nums.bin"

__global__ void add_vectors(double *a, double *b, double *c, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(tid < n){
        c[tid] = a[tid] + b[tid];
    }
    
}

void init_array(double *a, double *b){

    FILE *fptr = fopen(FILENAME, "rb");
    if(fptr == NULL){
        printf("Error opening file\n");
        exit(1);
    }


    for(size_t i = 0; i < N; i++) {
        fscanf(fptr, "%lf %lf", &a[i], &b[i]);
    }


    fclose(fptr);

}

double serial_add(double *a, double *b, double *c){
    
    clock_t start = clock();
    for(size_t i = 0; i < N; i++){
        c[i] = a[i] + b[i];
    }
    clock_t end = clock();

    double time = (double) (end - start) / CLOCKS_PER_SEC;

    printf("Time taken by serial code: %lf\n", time);
    return time;

}

void verify(double *a, double *b, double *c){
    for(size_t i = 0; i < N; i++){
        assert(c[i] == a[i] + b[i]);
    }
    printf("All elements are correct\n");
}

int main(void){

    size_t bytes = N * sizeof(double);

    double *a, *b, *c;
    double *dev_a, *dev_b, *dev_c;
    
    hipMalloc((void **) &dev_a, bytes);
    hipMalloc((void **) &dev_b, bytes);
    hipMalloc((void **) &dev_c, bytes);

    a = (double *) malloc(bytes);
    b = (double *) malloc(bytes);
    c = (double *) malloc(bytes);

    init_array(a, b);

    hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, bytes, hipMemcpyHostToDevice);


    double serial_time = serial_add(a, b, c);


    //Initialize the grid and block dimensions of the kernel
    int threads = 512;
    int blocks = (N + threads - 1) / threads;  

    clock_t start = clock();

    add_vectors<<<blocks, threads>>>(dev_a, dev_b, dev_c, N);
    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, bytes, hipMemcpyDeviceToHost);


    clock_t end = clock();

    double parallel_time = (double) (end - start) / CLOCKS_PER_SEC;

    printf("Time taken by GPU code: %lf\n", parallel_time);

    verify(a, b, c);

    double speedup = serial_time / parallel_time;
    printf("Speedup: %lf\n", speedup);

    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    

    return 0;
}
